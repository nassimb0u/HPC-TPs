
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 16
#define M 16

__global__ void prod(int **a, int *b, int *c) {
    const int t = 16;
    __shared__ int tmp[t];
    int i = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    int j = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    tmp[i] = a[i][j] * b[j] + a[i][j + 1] * b[j + 1];
    tmp[i + 1] = a[i + 1][j] * b[j] + a[i + 1][j + 1] * b[j + 1];
    __syncthreads();
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int k = 0; k < 4; k++) atomicAdd(&c[i + k], tmp[i + k]);
    }
}

int main() {
    int **a, *b, *c;
    int **d_a, *d_b, *d_c;
    dim3 grid(4, 4);
    dim3 block(2, 2);
    // variables pour le calcule du temps
    hipEvent_t start, stop;
    float elapsed_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // allocation
    a = (int **)malloc(N * sizeof(int *));
    for (int i = 0; i < N; i++) a[i] = (int *)malloc(M * sizeof(int));
    b = (int *)malloc(M * sizeof(int));
    c = (int *)malloc(N * sizeof(int));
    // init
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) a[i][j] = i == j;
    }
    for (int i = 0; i < M; i++) b[i] = i;
    // allocation mem GPU
    hipMalloc((void **)&d_a, N * sizeof(int *));
    for (int i = 0; i < N; i++) hipMalloc((void **)&d_a[i], M * sizeof(int));
    hipMalloc((void **)&d_b, M * sizeof(int));
    hipMalloc((void **)&d_c, N * sizeof(int));
    // cpy data
    for (int i = 0; i < N; i++)
        hipMemcpy(d_a[i], a[i], M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, M * sizeof(int), hipMemcpyHostToDevice);
    // call kernel
    hipEventRecord(start, 0);
    prod<<<grid, block>>>(d_a, d_b, d_c);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Elapsed time : %f ms\n", elapsed_time);
    // cpy results
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
    // show results
    printf("c = ");
    for (int i = 0; i < N; i++) printf("%d ", c[i]);
    printf("\n");
    // free mem gpu
    for (int i = 0; i < N; i++) hipFree(d_a[i]);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    // free mem
    for (int i = 0; i < N; i++) free(a[i]);
    free(a);
    free(b);
    free(c);
    return 0;
}
