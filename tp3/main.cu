
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 16
#define M 16

__global__ void prod(int *a, int *b, int *c) {
    int i = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    int j = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    int tmp1 = a[i * M + j] * b[j] + a[i * M + j + 1] * b[j + 1];
    int tmp2 = a[(i + 1) * M + j] * b[j] + a[(i + 1) * M + j + 1] * b[j + 1];
    atomicAdd(&c[i], tmp1);
    atomicAdd(&c[i + 1], tmp2);
}

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    dim3 grid(4, 4);
    dim3 block(2, 2);
    // variables pour le calcule du temps
    hipEvent_t start, stop;
    float elapsed_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // allocation
    a = (int *)malloc(N * M * sizeof(int));
    b = (int *)malloc(M * sizeof(int));
    c = (int *)malloc(N * sizeof(int));
    // init
    for (int i = 0; i < N * M; i++) a[i] = 1;
    for (int i = 0; i < M; i++) b[i] = i;
    // allocation mem GPU
    hipMalloc((void **)&d_a, N * M * sizeof(int));
    hipMalloc((void **)&d_b, M * sizeof(int));
    hipMalloc((void **)&d_c, N * sizeof(int));
    // cpy data
    hipMemcpy(d_a, a, N * M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, M * sizeof(int), hipMemcpyHostToDevice);
    // call kernel
    hipEventRecord(start, 0);
    prod<<<grid, block>>>(d_a, d_b, d_c);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Elapsed time : %f ms\n", elapsed_time);
    // cpy results
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
    // show results
    printf("c = ");
    for (int i = 0; i < N; i++) printf("%d ", c[i]);
    printf("\n");
    // free mem gpu
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    // free mem
    free(a);
    free(b);
    free(c);
    return 0;
}
