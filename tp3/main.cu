#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N 16
#define M 16

__global__ void prod(int *a, int *b, int *c) {
    const int t = 16;
    __shared__ int tmp[t];
    int i = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    int j = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    tmp[i] = a[i * M + j] * b[j] + a[i * M + j + 1] * b[j + 1];
    tmp[i + 1] = a[(i + 1) * M + j] * b[j] + a[(i + 1) * M + j + 1] * b[j + 1];
    __syncthreads();
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int k = 0; k < 4; k++) atomicAdd(&c[i + k], tmp[i + k]);
    }
}

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    dim3 grid(4, 4);
    dim3 block(2, 2);
    // variables pour le calcule du temps
    hipEvent_t start, stop;
    float elapsed_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // allocation
    a = (int *)malloc(N * M * sizeof(int));
    b = (int *)malloc(M * sizeof(int));
    c = (int *)malloc(N * sizeof(int));
    // init
    for (int i = 0; i < N; i++) a[i][j] = 1;
    for (int i = 0; i < M; i++) b[i] = i;
    // allocation mem GPU
    hipMalloc((void **)&d_a, N * M * sizeof(int));
    hipMalloc((void **)&d_b, M * sizeof(int));
    hipMalloc((void **)&d_c, N * sizeof(int));
    // cpy data
    hipMemcpy(d_a, a, M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, M * sizeof(int), hipMemcpyHostToDevice);
    // call kernel
    hipEventRecord(start, 0);
    prod<<<grid, block>>>(d_a, d_b, d_c);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Elapsed time : %f ms\n", elapsed_time);
    // cpy results
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
    // show results
    printf("c = ");
    for (int i = 0; i < N; i++) printf("%d ", c[i]);
    printf("\n");
    // free mem gpu
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    // free mem
    free(a);
    free(b);
    free(c);
    return 0;
}
